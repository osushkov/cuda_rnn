#include "hip/hip_runtime.h"

#include "ErrorMeasureKernel.cuh"
#include "Constants.hpp"
#include "../Types.cuh"
#include <hip/hip_runtime.h>
#include <cassert>

using namespace rnn;
using namespace rnn::cuda;

__global__
void errorMeasureKernel(ConnectionActivation nnOut, TargetOutput target, LayerBatchDeltas out) {
  const unsigned row = blockDim.y * blockIdx.y + threadIdx.y;
  const unsigned col = blockDim.x * blockIdx.x + threadIdx.x;

  if (row >= out.batchSize || col >= out.delta.cols) {
    return;
  }

  *Elem(out.delta, row, col) = *Elem(nnOut.activation, row, col) - *Elem(target.value, row, col);
}

void ErrorMeasureKernel::Apply(ConnectionActivation networkOutput, TargetOutput targetOutput,
                               LayerBatchDeltas out, hipStream_t stream) {

  assert(networkOutput.activation.cols == targetOutput.value.cols + 1);
  assert(out.delta.cols == targetOutput.value.cols);

  int bpgX = (out.delta.cols + TPB_X - 1) / TPB_X;
  int bpgY = (out.batchSize + TPB_Y - 1) / TPB_Y;

  errorMeasureKernel<<<dim3(bpgX, bpgY, 1), dim3(TPB_X, TPB_Y, 1), 0, stream>>>(
      networkOutput, targetOutput, out);
}
