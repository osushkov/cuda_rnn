
#include "Util.hpp"
#include "Util.cuh"
#include <hip/hip_runtime.h>
#include <cassert>

using namespace rnn;
using namespace rnn::cuda;

void *util::AllocPinned(size_t bufSize) {
  void* result = nullptr;

  hipError_t err = hipHostAlloc(&result, bufSize, hipHostMallocDefault);
  CheckError(err);
  assert(result != nullptr);

  return result;
}

void util::FreePinned(void *buf) {
  assert(buf != nullptr);
  hipError_t err = hipHostFree(buf);
  CheckError(err);
}

CuMatrix util::AllocMatrix(unsigned rows, unsigned cols) {
  CuMatrix result;

  result.data = nullptr;
  result.rows = rows;
  result.cols = cols;

  size_t width = cols * sizeof(float);
  size_t height = rows;

  hipError_t err = hipMallocPitch(&(result.data), &(result.pitch), width, height);
  CheckError(err);
  assert(result.data != nullptr);

  return result;
}

void util::FreeMatrix(CuMatrix &m) {
  assert(m.data != nullptr);
  hipError_t err = hipFree(m.data);
  CheckError(err);
  m.data = nullptr;
}
